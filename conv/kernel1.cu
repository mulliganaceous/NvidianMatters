#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include "kernels.h"

#include <stdio.h>
#include <string>

#define IDROWCOL                                     \
	int idx = blockIdx.x * blockDim.x + threadIdx.x; \
	int row = idx % height;                          \
	int col = idx / height;

void run_kernel1(const int8_t *filter, int32_t dimension,
				 const int32_t *input, int32_t *output,
				 int32_t width, int32_t height, int32_t smallest, int32_t largest)
{
	// Setting for single-thread-pixel
	const unsigned THREADS_PER_BLOCK = 512;
	dim3 threads(THREADS_PER_BLOCK, 1);
	dim3 blocks((width*height + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK);

	// launch kernel
	kernel1<<<blocks, threads>>>(filter, dimension, input, output, width, height);
	normalize1<<<blocks, threads>>>(output, width, height, smallest, largest);
}

__global__ void kernel1(const int8_t *filter, int32_t dimension,
						const int32_t *input, int32_t *output,
						int32_t width, int32_t height)
{
	// Thread identifier.
	IDROWCOL
	int32_t pix = 0;

	// Perform (no reduction needed). Each thread processes a pixel individually.
	if (idx < width * height) {
		// Perform filtering
		for (int ip = 0; ip < dimension; ip++) {
			for (int jp = 0; jp < dimension; jp++) {
				int32_t row_cur = row + ip - (dimension - 1) / 2;
				int32_t col_cur = col + jp - (dimension - 1) / 2;
				if (row_cur >= 0 && row_cur < height && col_cur >= 0 && col_cur < width) {
					pix += filter[dimension * ip + jp] * input[width * row_cur + col_cur];
				}
			}
		}
		// Update to shared data (a pointer)
		output[width * row + col] = pix;
	}
	__syncthreads();
}

__global__ void normalize1(int32_t *image, int32_t width, int32_t height,
						   int32_t smallest, int32_t biggest)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (smallest == biggest || idx >= width * height)
		return;
	if (idx < width * height) {
		image[idx] = ((image[idx] - smallest)*255) / (biggest - smallest);
	}
	__syncthreads();
}
