#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include "kernels.h"

__global__ void dot_kernel3(float *g_idata1, float *g_idata2, float *g_odata) {
    // Shared memory
    extern __shared__ int sdata[];
    
    // tid and global id counters
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    // Perform
    sdata[tid] = g_idata1[i]*g_idata2[i];
    __syncthreads();

    // Do reduction in shared memory
	for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) { 
        if (tid < s) {  
        sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block back to global memory
    if (tid == 0) { g_odata[blockIdx.x] = sdata[0]; }
}
